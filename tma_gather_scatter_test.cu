#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <random>

#include <hip/hip_runtime.h>
#include <thrust/sequence.h>
#include <thrust/reverse.h>
#include <thrust/execution_policy.h>
#include <thrust/sort.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <cuda/barrier>
#include <cuda/pipeline>

using namespace std;

typedef uint32_t InputT;
typedef uint32_t OutputT;
typedef uint32_t EmbeddingT;
typedef int IndexT;

#define SCATTER
//#define GATHER

#ifdef SCATTER
//#define QUICK_VALIDATION_SCATTER
#define shm_size (24576/sizeof(InputT))//this setting assure best performance for 1024xint feature
#endif
#ifdef GATHER
//#define QUICK_VALIDATION_GATHER
#define shm_size (16384/sizeof(InputT))//TODO this may be important, can be fine-tuned
                                       //in our experiments, less shm size may be better
#endif

//#define NAIVE_COPY
#define TMA_COPY
//#define TMA_PIPELINE_COPY

//#define shm_size (49152/sizeof(InputT))

struct desc{
  int size;
  int dim;
  int stride;
  int start_off;
  desc(int _s, int _d, int _stride, int _startoff):
        size(_s), dim(_d), stride(_stride), start_off(_startoff){}
};

#define CUDA_TRY(call)                                                          \
  do {                                                                          \
    hipError_t const status = (call);                                          \
    if (hipSuccess != status) {                                                \
      printf("%s %s %d\n", hipGetErrorString(status), __FILE__, __LINE__);  \
    }                                                                           \
  } while (0)

template <typename DataTypeT>
__device__ __forceinline__ void mov_typed_data(DataTypeT* to, const DataTypeT* from)
{
  *to = *from;
}
template <int DATA_SIZE>
__device__ __forceinline__ void mov_data(void* to, const void* from)
{
  char* ptr_to         = static_cast<char*>(to);
  const char* ptr_from = static_cast<const char*>(from);
  for (int i = 0; i < DATA_SIZE; i++) {
    ptr_to[i] = ptr_from[i];
  }
}
template <typename DataTypeT, int DATA_SIZE>
struct typed_data_vector {
  DataTypeT data[DATA_SIZE];
};
template <>
struct typed_data_vector<double, 2> {
  double2 data;
};
template <>
struct typed_data_vector<int64_t, 2> {
  int4 data;
};
template <>
struct typed_data_vector<float, 2> {
  float2 data;
};
template <>
struct typed_data_vector<float, 4> {
  float4 data;
};
template <>
struct typed_data_vector<int, 2> {
  int2 data;
};
template <>
struct typed_data_vector<int, 4> {
  int4 data;
};//WARNING: A BUG MAYBE
template <>
struct typed_data_vector<int16_t, 2> {
  int data;
};
template <>
struct typed_data_vector<int16_t, 4> {
  int2 data;
};
template <>
struct typed_data_vector<int16_t, 8> {
  int4 data;
};
template <>
struct typed_data_vector<int8_t, 2> {
  int16_t data;
};
template <>
struct typed_data_vector<int8_t, 4> {
  int data;
};
template <>
struct typed_data_vector<int8_t, 8> {
  int2 data;
};
template <>
struct typed_data_vector<int8_t, 16> {
  int4 data;
};
template <typename DataTypeT, int DATA_SIZE>
__device__ __forceinline__ DataTypeT& typed_data_vector_at(
  typed_data_vector<DataTypeT, DATA_SIZE>& v, int idx)
{
  return ((DataTypeT*)(&v.data))[idx];
}

template <>
__device__ __forceinline__ void mov_data<1>(void* to, const void* from)
{
  mov_typed_data(static_cast<int8_t*>(to), static_cast<const int8_t*>(from));
}
template <>
__device__ __forceinline__ void mov_data<2>(void* to, const void* from)
{
  mov_typed_data(static_cast<int16_t*>(to), static_cast<const int16_t*>(from));
}
template <>
__device__ __forceinline__ void mov_data<4>(void* to, const void* from)
{
  mov_typed_data(static_cast<int32_t*>(to), static_cast<const int32_t*>(from));
}
template <>
__device__ __forceinline__ void mov_data<8>(void* to, const void* from)
{
  mov_typed_data(static_cast<int64_t*>(to), static_cast<const int64_t*>(from));
}
template <>
__device__ __forceinline__ void mov_data<16>(void* to, const void* from)
{
  mov_typed_data(static_cast<int4*>(to), static_cast<const int4*>(from));
}

template <typename DataTypeT>
class type_caster {
 public:
  using LoadTypeT  = DataTypeT;
  using StoreTypeT = DataTypeT;
  static __device__ __forceinline__ LoadTypeT convert_load_data(DataTypeT data)
  {
    return static_cast<LoadTypeT>(data);
  }
  static __device__ __forceinline__ DataTypeT convert_store_data(StoreTypeT data)
  {
    return static_cast<DataTypeT>(data);
  }
};
template <>
class type_caster<__half> {
 public:
  using LoadTypeT  = float;
  using StoreTypeT = float;
  static __device__ __forceinline__ LoadTypeT convert_load_data(__half data)
  {
    return static_cast<LoadTypeT>(data);
  }
  static __device__ __forceinline__ __half convert_store_data(StoreTypeT data)
  {
    return static_cast<__half>(data);
  }
};


template <typename FromT, typename ToT>
__device__ __forceinline__ ToT convert_type(FromT from)
{
  return type_caster<ToT>::convert_store_data(type_caster<FromT>::convert_load_data(from));
}

/**
 * Determine alignment of a WholeMemory matrix, in element count, maximum 16 / element_size.
 * @param embedding_desc : wholememory_matrix_description_t matrix description.
 * @return : Alignment that can be used, in element count.
 */
inline int determine_wholememory_alignment_elt_count(desc embedding_desc)
{
  int elt_size = static_cast<int>(sizeof(EmbeddingT));
  assert(elt_size != -1);
  int alignment = 16 / elt_size;
  for (; alignment > 1; alignment /= 2) {
    if (embedding_desc.start_off % alignment == 0 &&
        embedding_desc.dim % alignment == 0 && embedding_desc.stride % alignment == 0)
      break;
  }
  return alignment;
}

/**
 * Determine alignment of normal memory, in element count, maximum 16 / element_size.
 * @param ptr : pointer to the memory.
 * @param memory_desc : wholememory_matrix_description_t matrix description.
 * @return : Alignment that can be used, in element count.
 */
inline int determine_memory_alignment_elt_count(const void* ptr, desc memory_desc)
{
  int elt_size = static_cast<int>(sizeof(InputT));
  assert(elt_size != -1);
  int alignment   = 16 / elt_size;
  int64_t int_ptr = reinterpret_cast<int64_t>(ptr);
  assert(int_ptr % elt_size == 0);
  int_ptr /= elt_size;
  int_ptr += memory_desc.start_off;
  for (; alignment > 1; alignment /= 2) {
    if (int_ptr % alignment == 0 && memory_desc.dim % alignment == 0 &&
        memory_desc.stride % alignment == 0)
      break;
  }
  return alignment;
}

template <int ALIGNMENT = 3>
__global__ void scatter_func_kernel(const InputT* input,
                                    desc input_desc,
                                    const IndexT* indices,
                                    int indice_count,
                                    EmbeddingT* embedding,
                                    desc embedding_desc)
{
  int64_t input_idx          = static_cast<int64_t>(blockIdx.x) * blockDim.y + threadIdx.y;
  int thread_idx             = threadIdx.x;
  IndexT embedding_table_idx = indices[input_idx];
  if (embedding_table_idx < 0) return;
  //wholememory::device_reference<EmbeddingT> embedding_dev_ref(embedding_gref);
  int embedding_size       = embedding_desc.dim;
  int64_t embedding_stride = embedding_desc.stride;
  int64_t input_stride     = input_desc.stride;
  typed_data_vector<EmbeddingT, ALIGNMENT> embeddings;
  typed_data_vector<InputT, ALIGNMENT> inputs;
  const InputT* input_ptr  = input + input_desc.start_off + input_stride * input_idx;
  int64_t embedding_offset = embedding_desc.start_off + embedding_table_idx * embedding_stride;
  for (; input_idx < indice_count; input_idx += static_cast<int64_t>(gridDim.x) * blockDim.y) {
    for (int emb_idx = thread_idx * ALIGNMENT; emb_idx < embedding_size; emb_idx += ALIGNMENT * blockDim.x) {
      mov_data<sizeof(InputT) * ALIGNMENT>(&inputs, input_ptr + emb_idx);
#pragma unroll
      for (int sub_idx = 0; sub_idx < ALIGNMENT; sub_idx++) {
        typed_data_vector_at(embeddings, sub_idx) =
          convert_type<InputT, EmbeddingT>(typed_data_vector_at(inputs, sub_idx));
      }
      mov_data<sizeof(EmbeddingT) * ALIGNMENT>(embedding + embedding_offset + emb_idx,
                                               &embeddings);
    }
  }
}

__inline__ __device__
void cp_async_bulk_global_to_shared(void *__dest, const void *__src, _CUDA_VSTD::uint32_t __size, ::cuda::barrier<::cuda::thread_scope_block> &__bar)
{
    assert(__size % 16 == 0);
    assert(__isShared(__dest));
    assert(__isGlobal(__src));

    asm volatile(
        "cp.async.bulk.shared::cluster.global.mbarrier::complete_tx::bytes [%0], [%1], %2, [%3];\n"
        :
        : "r"(static_cast<_CUDA_VSTD::uint32_t>(__cvta_generic_to_shared(__dest))),
          "l"(static_cast<_CUDA_VSTD::uint64_t>(__cvta_generic_to_global(__src))),
          "r"(__size),
          "r"(static_cast<_CUDA_VSTD::uint32_t>(__cvta_generic_to_shared(::cuda::device::barrier_native_handle(__bar))))
        : "memory");
}

template <int ALIGNMENT = 3>
__global__ void scatter_kernel_TMA(const InputT* input,
                                    desc input_desc,
                                    const IndexT* indices,
                                    int indice_count,
                                    EmbeddingT* embedding,
                                    desc embedding_desc)
{
  
  auto block = cooperative_groups::this_thread_block();
  auto mywarp = cooperative_groups::tiled_partition<32>(block);
  extern __shared__ InputT shared[];
  InputT* my_shared;
  int warp_id = (threadIdx.x + blockIdx.x * blockDim.x)/32;
  int lane_id = threadIdx.x % 32;

  using barrier = cuda::barrier<cuda::thread_scope_block>;
  __shared__ barrier bar;
  if (threadIdx.x == 0) {
    init(&bar, blockDim.x);
  }
  block.sync();

  int embedding_size       = embedding_desc.dim;
  int64_t embedding_stride = embedding_desc.stride;
  int block_idx = block.group_index().x;
  int64_t input_stride     = input_desc.stride;
  int async_copy_align = 16;
  int batch_size = (shm_size/(blockDim.x/32)-async_copy_align)/input_stride;//indices batch size in lines
  typed_data_vector<EmbeddingT, ALIGNMENT> embeddings;
  typed_data_vector<InputT, ALIGNMENT> inputs;
  int input_off_tail = input_desc.start_off%async_copy_align;//this is crutial for copy alignment, 4 bytes as alignment;
  bool use_shm = true;
  if (batch_size <= 0) { 
    use_shm = false; batch_size = 1;
  }else {
    my_shared = shared + shm_size/(blockDim.x/32)*(threadIdx.x/32);
  }
  for (int64_t input_idx = warp_id*batch_size; input_idx < indice_count; input_idx += gridDim.x*(blockDim.x/32)*batch_size) {
	  int cur_idx_lines = (indice_count - input_idx) > batch_size ? batch_size : indice_count - input_idx;
	  const InputT* input_ptr = input + input_desc.start_off - input_off_tail + input_stride * input_idx;
    //this variable is also for alignment
    if (use_shm) {
      int copy_size = (input_off_tail + cur_idx_lines*input_stride)*sizeof(InputT);
      if (input_idx + cur_idx_lines < indice_count)//input_dim * sizeof(InputT) > 4 is needed
        copy_size = (copy_size + async_copy_align -1)/async_copy_align*async_copy_align;
      cp_async_bulk_global_to_shared((void *)my_shared, (void *)input_ptr, copy_size, bar);
	    //cooperative_groups::memcpy_async(mywarp, my_shared, input_ptr, copy_size);
	    //cooperative_groups::wait(mywarp);

    }
	  for (int e = 0; e < cur_idx_lines; e ++) {
		  int64_t embedding_table_idx = indices[input_idx + e];
	  	EmbeddingT *emb_ptr = embedding + embedding_desc.start_off + embedding_table_idx*embedding_stride;
      
      for (int emb_idx = lane_id * ALIGNMENT; emb_idx < embedding_size; emb_idx += ALIGNMENT * 32) {
        if (use_shm) mov_data<sizeof(InputT) * ALIGNMENT>(&inputs, my_shared + input_off_tail + e*input_stride + emb_idx);
        else mov_data<sizeof(InputT) * ALIGNMENT>(&inputs, input_ptr + input_off_tail + e*input_stride + emb_idx);
#pragma unroll
        for (int sub_idx = 0; sub_idx < ALIGNMENT; sub_idx++) {
          typed_data_vector_at(embeddings, sub_idx) =
            convert_type<InputT, EmbeddingT>(typed_data_vector_at(inputs, sub_idx));
        }
        mov_data<sizeof(EmbeddingT) * ALIGNMENT>(emb_ptr + emb_idx, &embeddings);
      }
	  }
    mywarp.sync();
  }
  return ;
}


template<int ALIGNMENT = 3,uint8_t stage_count = 2>//TODO set stage count to 2
__global__ void scatter_kernel_TMA_pipeline(const InputT* input,
                                    desc input_desc,
                                    const IndexT* indices,
                                    int indice_count,
                                    EmbeddingT* embedding,
                                    desc emb_desc)
{
  //auto grid = cooperative_groups::this_grid();
  auto block = cooperative_groups::this_thread_block();

  extern __shared__ EmbeddingT shared[]; // stages_count * block.size() * sizeof(int) bytes
  
  __shared__ cuda::pipeline_shared_state<cuda::thread_scope::thread_scope_block,stage_count> shared_state;
  auto pipeline = cuda::make_pipeline(block, &shared_state);

  int input_stride = input_desc.stride;
  int batch_size = shm_size/stage_count/input_stride;
  int block_idx = block.group_index().x;
  int my_batch_num = ((indice_count+batch_size-1)/batch_size) % gridDim.x > block_idx ? 1 : 0;
  my_batch_num += indice_count/(batch_size*gridDim.x);
  
  typed_data_vector<EmbeddingT, ALIGNMENT> embeddings;
  typed_data_vector<InputT, ALIGNMENT> inputs;

  for (int get_batch = 0, put_batch = 0; put_batch < my_batch_num; put_batch ++) {
    for (; get_batch < my_batch_num && get_batch < (put_batch + stage_count); get_batch ++) {
      int64_t input_idx = (get_batch * gridDim.x + block_idx) * batch_size;
      int idx_line_num = (indice_count - input_idx) > batch_size ? batch_size : (indice_count - input_idx);
      int shared_off = (get_batch % stage_count)*batch_size*input_stride;
      EmbeddingT *in_addr = (EmbeddingT*)(input + input_desc.start_off + input_stride * input_idx);
      pipeline.producer_acquire();
      cuda::memcpy_async(block, shared+shared_off, in_addr, sizeof(EmbeddingT)*idx_line_num*input_stride, pipeline);
      pipeline.producer_commit();
    }
    int64_t input_idx = (put_batch * gridDim.x + block_idx) * batch_size;
    int idx_line_num = (indice_count - input_idx) > batch_size ? batch_size : (indice_count - input_idx);
    pipeline.consumer_wait();
    for (int e = 0; e < idx_line_num; e ++) {
      int64_t emb_idx = indices[input_idx+e];
      EmbeddingT* out_addr = embedding + emb_desc.start_off + emb_desc.stride*emb_idx;
      int shared_off = (put_batch % stage_count)* batch_size*input_stride;
      /*for (int emb_idx = block.thread_rank(); emb_idx < emb_desc.dim; emb_idx += block.size()) {
        out_addr[emb_idx] = shared[shared_off+e*input_stride+emb_idx];
      }*/
      for (int emb_idx = block.thread_rank() * ALIGNMENT; emb_idx < emb_desc.dim; emb_idx += ALIGNMENT * block.size()) {
        mov_data<sizeof(InputT) * ALIGNMENT>(&inputs, shared + shared_off + e*input_stride + emb_idx);
#pragma unroll
        for (int sub_idx = 0; sub_idx < ALIGNMENT; sub_idx++) {
          typed_data_vector_at(embeddings, sub_idx) =
            convert_type<InputT, EmbeddingT>(typed_data_vector_at(inputs, sub_idx));
        }
        mov_data<sizeof(EmbeddingT) * ALIGNMENT>(out_addr + emb_idx, &embeddings);
      }
    }
    //block.sync();
    pipeline.consumer_release();
    block.sync();
  }
  return ;
}

void scatter_temp_func(InputT* input,
                       struct desc input_desc,
                       IndexT* indices,
                       int indice_count,
                       EmbeddingT* embedding,
                       struct desc embedding_desc)
{
  if (indice_count == 0 || embedding_desc.dim == 0) return;
  int wm_alignment   = determine_wholememory_alignment_elt_count(embedding_desc);
  int mm_alignment   = determine_memory_alignment_elt_count(input, input_desc);
  int alignment      = std::min<int>(wm_alignment, mm_alignment);
  int embedding_size = embedding_desc.dim;
#ifdef NAIVE_COPY
  int thread_x       = (embedding_size + alignment-1)/alignment;
  thread_x           = std::min(thread_x, 256);
  int thread_y       = 1;
  if (thread_x < 64) {
    int power2_thread_x = 1;
    for (; power2_thread_x < thread_x; power2_thread_x *= 2)
      ;
    thread_x = power2_thread_x;
    thread_y = 64 / thread_x;
  }
  int64_t block_count_64 = (indice_count + thread_y - 1) / thread_y;
  int block_count = block_count_64 >= INT_MAX ? INT_MAX / 4 : static_cast<int>(block_count_64);
  dim3 block_dim(thread_x, thread_y, 1);
  void (*kernel_fn)(const InputT*,
                    desc,
                    const IndexT*,
                    int,
                    EmbeddingT*,
                    desc) = nullptr;
  //printf("key parameters: %d %d %d %d\n",thread_x, thread_y, block_count, alignment);
  switch (alignment) {
    case 16: {kernel_fn = scatter_func_kernel<16>;break;}
    case 8: {kernel_fn = scatter_func_kernel<8>;break;}
    case 4: {kernel_fn = scatter_func_kernel<4>;break;}
    case 2: {kernel_fn = scatter_func_kernel<2>;break;}
    case 1: {kernel_fn = scatter_func_kernel<1>;break;}
    default: {
      printf("scatter func alignment=%d.\n", alignment);
      return;
    }
  }
#endif
  hipEvent_t start, stop;
	float esp_time_gpu;
	CUDA_TRY(hipEventCreate(&start));
	CUDA_TRY(hipEventCreate(&stop));
  CUDA_TRY(hipEventRecord(start, 0));
#ifdef NAIVE_COPY
  kernel_fn<<<block_count, block_dim>>>(input,
                                        input_desc,
                                        indices,
                                        indice_count,
                                        embedding,
                                        embedding_desc);
#endif
#ifdef TMA_COPY
  void (*kernel_fn)(const InputT*,
                    desc,
                    const IndexT*,
                    int,
                    EmbeddingT*,
                    desc) = nullptr;
  //printf("key parameters: %d %d %d %d\n",thread_x, thread_y, block_count, alignment);
  switch (alignment) {
    case 16: {  kernel_fn = scatter_kernel_TMA<16>; break;}
    case 8: {  kernel_fn = scatter_kernel_TMA<8>; break;}
    case 4: {  kernel_fn = scatter_kernel_TMA<4>; break;}
    case 2: {  kernel_fn = scatter_kernel_TMA<2>; break;}
    case 1: {  kernel_fn = scatter_kernel_TMA<1>; break;}
    default: {
      printf("scatter func alignment=%d.\n", alignment); return;
    }
  }
  int block_size = (embedding_desc.dim + alignment-1)/alignment;
  //block_size = block_size > 512 ? 512 : block_size;
  block_size = 256;
  int block_count = indice_count > 2048 ? 2048 : indice_count;
  kernel_fn<<<block_count, block_size, shm_size*sizeof(InputT)>>>(input,
                                                                          input_desc,
                                                                          indices,
                                                                          indice_count,
                                                                          embedding,
                                                                          embedding_desc);
#endif
#ifdef TMA_PIPELINE_COPY
   void (*kernel_fn)(const InputT*,
                    desc,
                    const IndexT*,
                    int,
                    EmbeddingT*,
                    desc) = nullptr;
  //printf("key parameters: %d %d %d %d\n",thread_x, thread_y, block_count, alignment);
  switch (alignment) {
    case 16: {  kernel_fn = scatter_kernel_TMA_pipeline<16,2>; break;}
    case 8: {  kernel_fn = scatter_kernel_TMA_pipeline<8,2>; break;}
    case 4: {  kernel_fn = scatter_kernel_TMA_pipeline<4,2>; break;}
    case 2: {  kernel_fn = scatter_kernel_TMA_pipeline<2,2>; break;}
    case 1: {  kernel_fn = scatter_kernel_TMA_pipeline<1,2>; break;}
    default: {
      printf("scatter func alignment=%d.\n", alignment); return;
    }
  }
  int block_size = (embedding_desc.dim + alignment-1)/alignment;
  block_size = block_size > 256 ? 256 : block_size;
  int block_count = indice_count > 4096 ? 4096 : indice_count;
  kernel_fn<<<block_count, block_size, shm_size*sizeof(EmbeddingT)>>>(input,
                                                                   input_desc,
                                                                   indices,
                                                                   indice_count,
                                                                   embedding,
                                                                   embedding_desc);

#endif
  CUDA_TRY(hipDeviceSynchronize());
  CUDA_TRY(hipEventRecord(stop, 0));
	CUDA_TRY(hipEventSynchronize(stop));
  CUDA_TRY(hipEventElapsedTime(&esp_time_gpu, start, stop));
	printf("    Time for the kernel is: %f ms, where alignment is %d\n", esp_time_gpu, alignment);
  return ;
}

template <int ALIGNMENT = 1>
__global__ void gather_func_kernel(EmbeddingT* embedding,
                                   desc embedding_desc,
                                   const IndexT* indices,
                                   int64_t indice_count,
                                   OutputT* output,
                                   desc output_desc)
{
  int64_t output_idx         = static_cast<int64_t>(blockIdx.x) * blockDim.y + threadIdx.y;
  IndexT embedding_table_idx = indices[output_idx];
  if (embedding_table_idx < 0) return;
  int thread_idx           = threadIdx.x;
  int embedding_size       = embedding_desc.dim;
  int64_t embedding_stride = embedding_desc.stride;
  int64_t output_stride    = output_desc.stride;
  typed_data_vector<EmbeddingT, ALIGNMENT> embeddings;
  typed_data_vector<OutputT, ALIGNMENT> outputs;
  OutputT* output_ptr      = output + output_desc.start_off + output_stride * output_idx;
  int64_t embedding_offset = embedding_desc.start_off + embedding_table_idx * embedding_stride;
  for (; output_idx < indice_count; output_idx += static_cast<int64_t>(gridDim.x) * blockDim.y) {
    for (int emb_idx = thread_idx * ALIGNMENT; emb_idx < embedding_size;
         emb_idx += ALIGNMENT * blockDim.x) {
      mov_data<sizeof(EmbeddingT) * ALIGNMENT>(&embeddings, embedding + embedding_offset + emb_idx);
#pragma unroll
      for (int sub_idx = 0; sub_idx < ALIGNMENT; sub_idx++) {
        typed_data_vector_at(outputs, sub_idx) =
          convert_type<EmbeddingT, OutputT>(typed_data_vector_at(embeddings, sub_idx));
      }
      mov_data<sizeof(OutputT) * ALIGNMENT>(output_ptr + emb_idx, &outputs);
    }
  }
}

template <int ALIGNMENT = 1>
__global__ void gather_func_kernel_TMA(EmbeddingT* embedding,
                                   desc embedding_desc,
                                   const IndexT* indices,
                                   int64_t indice_count,
                                   OutputT* output,
                                   desc output_desc)
{
  auto block = cooperative_groups::this_thread_block();
  auto mywarp = cooperative_groups::tiled_partition<32>(block);
  extern __shared__ OutputT shared[];
  OutputT* my_shared; 
  int warp_id = (threadIdx.x + blockIdx.x * blockDim.x)/32;
  int lane_id = threadIdx.x % 32;

  int embedding_size       = embedding_desc.dim;
  int64_t embedding_stride = embedding_desc.stride;
  int block_idx = block.group_index().x;
  int64_t output_stride     = output_desc.stride;
  //int async_copy_align = 4;
  int batch_size = shm_size/(blockDim.x/32)/output_stride;//indices batch size for a block in lines
  
  typed_data_vector<EmbeddingT, ALIGNMENT> embeddings;
  typed_data_vector<OutputT, ALIGNMENT> outputs;

  bool use_shm = true;
  if (batch_size <= 0) {
    use_shm = false;
    batch_size = 1;
  } else {
    my_shared = shared + shm_size/(blockDim.x/32)*(threadIdx.x/32);
  }
  //int output_off_tail = output_desc.storage_offset%async_copy_align;//this is crutial for copy alignment, 4 bytes as alignment;
  
  for (int64_t output_idx = warp_id*batch_size; output_idx < indice_count; output_idx += gridDim.x*(blockDim.x/32)*batch_size) {
	  int cur_idx_lines = (indice_count - output_idx) > batch_size ? batch_size : indice_count - output_idx;
    OutputT* output_ptr = output + output_desc.start_off + output_stride * output_idx;
    if (!use_shm) {
      my_shared = output_ptr;
    }
    for (int e = 0; e < cur_idx_lines; e ++) {
		  int64_t embedding_table_idx = indices[output_idx + e];
	  	EmbeddingT *emb_ptr = embedding + embedding_desc.start_off + embedding_table_idx*embedding_stride;
      
      for (int emb_idx = lane_id * ALIGNMENT; emb_idx < embedding_size; emb_idx += ALIGNMENT * 32) {
        mov_data<sizeof(EmbeddingT) * ALIGNMENT>(&embeddings, emb_ptr + emb_idx);
#pragma unroll
        for (int sub_idx = 0; sub_idx < ALIGNMENT; sub_idx++) {
          typed_data_vector_at(outputs, sub_idx) =
            convert_type<EmbeddingT, OutputT>(typed_data_vector_at(embeddings, sub_idx));
        }
        mov_data<sizeof(InputT) * ALIGNMENT>(my_shared + e*output_stride + emb_idx, &outputs);
      }
	  }
    //block.sync();
    if (use_shm) {
      //this variable is also for alignment
      int copy_size = cur_idx_lines*output_stride*sizeof(OutputT);
      //if (input_idx + cur_idx_lines < indice_count)//input_dim * sizeof(InputT) > 4 is needed
      //  copy_size = (copy_size + async_copy_align - 1)/async_copy_align*async_copy_align;
	    cooperative_groups::memcpy_async(mywarp, output_ptr, my_shared, copy_size);
	    cooperative_groups::wait(mywarp);
    } 
  }
  return;
}


void gather_temp_func(EmbeddingT *embedding,
                      desc embedding_desc,
                      IndexT* indices,
                      int64_t indice_count,
                      OutputT* output,
                      desc output_desc)
{
  int wm_alignment   = determine_wholememory_alignment_elt_count(embedding_desc);
  int mm_alignment   = determine_memory_alignment_elt_count(output, output_desc);
  int alignment      = std::min<int>(wm_alignment, mm_alignment);
  int embedding_size = embedding_desc.dim;
#ifdef NAIVE_COPY
  int thread_x       = (embedding_size+alignment-1)/alignment*alignment;
  thread_x           = std::min(thread_x, 256);
  int thread_y       = 1;
  if (thread_x < 64) {
    int power2_thread_x = 1;
    for (; power2_thread_x < thread_x; power2_thread_x *= 2)
      ;
    thread_x = power2_thread_x;
    thread_y = 64 / thread_x;
  }
  int64_t block_count_64 = (indice_count + thread_y - 1) / thread_y;
  int block_count = block_count_64 >= INT_MAX ? INT_MAX / 4 : static_cast<int>(block_count_64);
  dim3 block_dim(thread_x, thread_y, 1);
  void (*kernel_fn)(EmbeddingT*,
                    desc,
                    const IndexT*,
                    int64_t,
                    OutputT*,
                    desc) = nullptr;
  switch (alignment) {
    case 16: { kernel_fn = gather_func_kernel<16>; break;}
    case 8: { kernel_fn = gather_func_kernel<8>; break;}
    case 4: { kernel_fn = gather_func_kernel<4>; break;}
    case 2: { kernel_fn = gather_func_kernel<2>; break;}
    case 1: { kernel_fn = gather_func_kernel<1>; break;}
    default: {
      printf("gather func alignment=%d.", alignment);
      return;
    }
  }
#endif
#ifdef TMA_COPY
  int thread_num = (embedding_size+alignment-1)/ alignment;
  //thread_num = std::min(thread_num, 512);
  thread_num = 256;
  int64_t block_count = indice_count >= 2048 ? 2048 : static_cast<int>(indice_count);
  
  void (*kernel_fn)(EmbeddingT*,
                    desc,
                    const IndexT*,
                    int64_t,
                    OutputT*,
                    desc) = nullptr;
  switch (alignment) {
    case 16: { kernel_fn = gather_func_kernel_TMA<16>; break;}
    case 8: { kernel_fn = gather_func_kernel_TMA<8>; break;}
    case 4: { kernel_fn = gather_func_kernel_TMA<4>; break;}
    case 2: { kernel_fn = gather_func_kernel_TMA<2>; break;}
    case 1: { kernel_fn = gather_func_kernel_TMA<1>; break;}
    default: {
      printf("gather func alignment=%d.", alignment);
      return;
    }
  }
  
#endif
  hipEvent_t start, stop;
	float esp_time_gpu;
	CUDA_TRY(hipEventCreate(&start));
	CUDA_TRY(hipEventCreate(&stop));
  CUDA_TRY(hipEventRecord(start, 0));
#ifdef NAIVE_COPY
  kernel_fn<<<block_count, block_dim>>>(embedding,
                                        embedding_desc,
                                        indices,
                                        indice_count,
                                        output,
                                        output_desc);
#endif
#ifdef TMA_COPY
  kernel_fn<<<block_count, thread_num, shm_size*sizeof(OutputT)>>>(embedding,
                                                                  embedding_desc,
                                                                  indices,
                                                                  indice_count,
                                                                  output,
                                                                  output_desc);
#endif
  CUDA_TRY(hipDeviceSynchronize());
  CUDA_TRY(hipEventRecord(stop, 0));
	CUDA_TRY(hipEventSynchronize(stop));
  CUDA_TRY(hipEventElapsedTime(&esp_time_gpu, start, stop));
	printf("    Time for the kernel is: %f ms, where alignment is %d\n", esp_time_gpu, alignment);
  CUDA_TRY(hipGetLastError());
}


int main (int argc, char**argv) {
  //key parameters
  int embedding_dim = 128;
  if (argc > 1) embedding_dim = atoi(argv[1]);
  int emb_start_off = 0;//the offset is also in element
  if (argc > 2) emb_start_off = atoi(argv[2]);
  int input_start_off = 0;//emb_start_off;
  int output_start_off = 0;//
  if (argc > 3) {
    input_start_off = atoi(argv[3]);
    output_start_off = atoi(argv[3]);
  }
  int output_dim = embedding_dim;
  int input_dim = embedding_dim;
  uint64_t embedding_size = 10 * 1024UL * 1024UL;
  uint64_t input_size = embedding_size/2;
  uint64_t output_size = embedding_size/2;
  printf("the embedding dim is %d, emb_start_off %d, input/output_start_off %d\n", embedding_dim, emb_start_off, input_start_off);

  uint64_t total_size_gb = (embedding_size + input_size)*embedding_dim*sizeof(EmbeddingT)/1024/1024/1024;
  printf("the total size is %d GB\n", total_size_gb);
#ifdef SCATTER
  //construct input
  InputT *input;
  int in_aligned_size = 16/sizeof(InputT);
  int in_stride = input_dim % in_aligned_size == 0 ? 
                     input_dim : (input_dim/in_aligned_size+1)*in_aligned_size;
  int64_t in_malloc_size = (int64_t)in_stride * input_size + input_start_off;
  CUDA_TRY(hipMalloc((void **)&input, sizeof(InputT)*in_malloc_size));
  //printf("the input stride is %d, the input_malloc_size is %ld, ptr is 0x%p\n", in_stride, in_malloc_size, input);

  thrust::sequence(thrust::device, input+input_start_off, input+in_malloc_size, 0);//NOTE: more initialization methods needed
  thrust::reverse(thrust::device, input+input_start_off, input+in_malloc_size);
  struct desc input_desc = desc(input_size, input_dim, in_stride, input_start_off);
  printf("construct input tensor done, the in_stride is %d\n", in_stride);
#endif

  //construct embedding
  EmbeddingT * embedding;
  int emb_aligned_size = 16/sizeof(EmbeddingT);
  int emb_stride = embedding_dim % emb_aligned_size == 0 ? 
                     embedding_dim : (embedding_dim/emb_aligned_size+1)*emb_aligned_size;
  int64_t emb_malloc_size = (int64_t)emb_stride * embedding_size + emb_start_off;
  CUDA_TRY(hipMalloc((void **)&embedding, sizeof(EmbeddingT)*emb_malloc_size));
  //printf("the emb stride is %d, the emb_malloc_size is %ld, the ptr is 0x%p\n", emb_stride, emb_malloc_size, embedding);

  thrust::sequence(thrust::device, embedding+emb_start_off, embedding+emb_malloc_size, 0);
  struct desc emb_desc = desc(embedding_size, embedding_dim, emb_stride, emb_start_off);
  printf("construct the target embedding done, the emb_stride is %d\n", emb_stride);

#ifdef GATHER
  //construct output;
  OutputT* output;
  int out_aligned_size = 16/sizeof(OutputT);
  int out_stride = output_dim % out_aligned_size == 0 ? 
                     output_dim : (output_dim/out_aligned_size+1)*out_aligned_size;
  int64_t out_malloc_size = (int64_t)out_stride * output_size + output_start_off;
  CUDA_TRY(hipMalloc((void **)&output, sizeof(OutputT)*out_malloc_size));
  //printf("the input stride is %d, the input_malloc_size is %ld, ptr is 0x%p\n", in_stride, in_malloc_size, input);
  struct desc output_desc = desc(output_size, output_dim, out_stride, output_start_off);
  printf("construct output tensor done, the out_stride is %d\n", out_stride);
#endif

  //construct indices
  IndexT *indices;
  CUDA_TRY(hipMalloc((void **)&indices, sizeof(IndexT)*input_size));
  IndexT *h_indices = (IndexT*)malloc(sizeof(IndexT)*input_size);
#ifdef QUICK_VALIDATION_SCATTER 
  EmbeddingT* h_embedding = (EmbeddingT*)malloc(sizeof(EmbeddingT)*emb_malloc_size);
  InputT* h_input = (InputT *)malloc(sizeof(InputT)*in_malloc_size);
#endif
#ifdef QUICK_VALIDATION_GATHER
  EmbeddingT* h_embedding = (EmbeddingT*)malloc(sizeof(EmbeddingT)*emb_malloc_size);
  OutputT* h_output = (OutputT *)malloc(sizeof(OutputT)*out_malloc_size);
#endif
  uint8_t* used = (uint8_t*)malloc(sizeof(uint8_t)*embedding_size); 
  for (int iter = 0; iter < 2; iter ++) {
    printf("    start generating the indices for %d th iteration...\n", iter);
    //different iterations have different indices
    int min = 0, max = embedding_size-1;
    random_device seed;
	  ranlux48 engine(seed());
    uniform_int_distribution<> distrib(min, max);
    memset(used, 0, sizeof(uint8_t)*embedding_size);
    for (int i = 0; i < input_size; i ++) {
      int random = distrib(engine);//随机数
      while(used[random]) {
        random = distrib(engine);
      }
      used[random] = 1;
      //NOTE: currently only int/half_int is supported
      h_indices[i] = (IndexT)random;
    }
    thrust::sort(thrust::host, h_indices, h_indices + input_size);
    CUDA_TRY(hipMemcpy(indices, h_indices, sizeof(IndexT)*input_size, hipMemcpyHostToDevice));
    CUDA_TRY(hipDeviceSynchronize());
    printf("    indices prepared, start the scatter function now...\n");
#ifdef SCATTER
    scatter_temp_func(input,
                      input_desc,
                      indices,
                      input_size,
                      embedding,
                      emb_desc);
#endif
#ifdef GATHER
    gather_temp_func(embedding,
                     emb_desc,
                     indices,
                     input_size,
                     output,
                     output_desc);
#endif
#ifdef QUICK_VALIDATION_GATHER
    CUDA_TRY(hipMemcpy(h_embedding, embedding, sizeof(EmbeddingT)*emb_malloc_size, hipMemcpyDeviceToHost));
    CUDA_TRY(hipMemcpy(h_indices, indices, sizeof(IndexT)*output_size, hipMemcpyDeviceToHost));
    CUDA_TRY(hipMemcpy(h_output, output, sizeof(OutputT)*out_malloc_size, hipMemcpyDeviceToHost));
    CUDA_TRY(hipDeviceSynchronize());
    bool passed = true;
    for (int64_t i = 0; i < input_size; i ++) {
      int64_t emb_idx = h_indices[i];
      if (h_output[i*out_stride+output_start_off] != emb_idx*emb_stride) {
        passed = false;
        printf("i = %lu, the first ele of output is %d, should be %d\n",h_output[i*out_stride+output_start_off], emb_idx*emb_stride);
        break;
      }
    }
    if (passed)
      printf("    the %d th iteration passed quick validation!\n", iter);
    else 
      printf("    the %d th iteration did NOT pass the quick validation!\n", iter);
#endif
#ifdef QUICK_VALIDATION_SCATTER//NOTE the check here is designed for int type
    CUDA_TRY(hipMemcpy(h_embedding, embedding, sizeof(EmbeddingT)*emb_malloc_size, hipMemcpyDeviceToHost));
    CUDA_TRY(hipMemcpy(h_indices, indices, sizeof(IndexT)*input_size, hipMemcpyDeviceToHost));
    CUDA_TRY(hipMemcpy(h_input, input, sizeof(InputT)*in_malloc_size, hipMemcpyDeviceToHost));
    CUDA_TRY(hipDeviceSynchronize());

    /*printf("the embedding table:\n");
    for (uint64_t i = 0; i < embedding_size; i ++) {
      for (int j = 0; j < embedding_dim; j ++)
        printf("%d ", h_embedding[i*emb_stride+j+emb_start_off]);
      printf("\n");
    }
    printf("the input table:\n");
    for (uint64_t i = 0; i < input_size; i ++) {
      for (int j = 0; j < input_dim; j ++)
        printf("%d ", h_input[i*in_stride+j+input_start_off]);
      printf("\n");
    }
    printf("the indices:\n");
    for (int i = 0; i < input_size; i ++)
      printf("%d ", h_indices[i]);
    printf("\n");*/

    int index_pos = 0;
    //int index_print_step = (input_size/100 > 1) ? input_size/100 : 1;
    bool valid = true;
    for (uint64_t i = 0; i < embedding_size; i ++) {
      if (i == h_indices[index_pos]) {
        //if (index_pos % index_print_step == 0)
          //printf("the %d th index for %d th iteration is %ld\n", index_pos, iter, i);
        if (h_embedding[i*emb_stride + emb_start_off] != in_malloc_size-input_start_off-1-index_pos*in_stride) {
          valid = false;
          printf("scattered, i = %lu, index_pos = %d, embedding ele is %d, should be %d\n",
                                i, index_pos, h_embedding[i*emb_stride + emb_start_off], 
                                in_malloc_size-input_start_off-1-index_pos*in_stride);
          break;
        }
        index_pos ++;
      } else {
        if (h_embedding[i*emb_stride + emb_start_off] != i*emb_stride) {
          valid = false;
          printf("original, i = %lu, embedding ele is %d, should be %lu\n",
                            i, h_embedding[i*emb_stride + emb_start_off], i*emb_stride);
          break;
        }
      }
    }
    if (valid)
      printf("    the %d th iteration passed the quick validation!\n", iter);
    else 
      printf("    the %d th iteration didn't pass!\n", iter);
#endif
#ifdef SCATTER
    thrust::sequence(thrust::device, embedding+emb_start_off, embedding+emb_malloc_size, 0);
#endif
    printf("\n");
  }
#ifdef QUICK_VALIDATION_SCATTER
  free(h_embedding);
  free(h_input);
#endif
#ifdef QUICK_VALIDATION_GATHER
  free(h_embedding);
  free(h_output);
#endif
  free(used);
  free(h_indices);
#ifdef SCATTER
  CUDA_TRY(hipFree(input));
#endif
#ifdef GATHER
  CUDA_TRY(hipFree(output));
#endif
  CUDA_TRY(hipFree(embedding));
  CUDA_TRY(hipFree(indices));
  printf("exit now\n");
  return 0;
}
