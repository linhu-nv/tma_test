#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <random>

#include <hip/hip_runtime.h>
#include <thrust/sequence.h>
#include <thrust/reverse.h>
#include <thrust/execution_policy.h>
#include <thrust/sort.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <cuda/barrier>
#include <cuda/pipeline>

using namespace std;
//#define QUICK_VALIDATION
//#define NAIVE_COPY
//#define TMA_COPY
#define TMA_PIPELINE_COPY
//#define USE_BARRIER

typedef uint32_t InputT;
typedef uint32_t EmbeddingT;
typedef int IndexT;
struct desc{
  int size;
  int dim;
  int stride;
  int start_off;
  desc(int _s, int _d, int _stride, int _startoff):
        size(_s), dim(_d), stride(_stride), start_off(_startoff){}
};

#define CUDA_TRY(call)                                                          \
  do {                                                                          \
    hipError_t const status = (call);                                          \
    if (hipSuccess != status) {                                                \
      printf("%s %s %d\n", hipGetErrorString(status), __FILE__, __LINE__);  \
    }                                                                           \
  } while (0)

template <typename DataTypeT>
__device__ __forceinline__ void mov_typed_data(DataTypeT* to, const DataTypeT* from)
{
  *to = *from;
}
template <int DATA_SIZE>
__device__ __forceinline__ void mov_data(void* to, const void* from)
{
  char* ptr_to         = static_cast<char*>(to);
  const char* ptr_from = static_cast<const char*>(from);
  for (int i = 0; i < DATA_SIZE; i++) {
    ptr_to[i] = ptr_from[i];
  }
}
template <typename DataTypeT, int DATA_SIZE>
struct typed_data_vector {
  DataTypeT data[DATA_SIZE];
};
template <>
struct typed_data_vector<double, 2> {
  double2 data;
};
template <>
struct typed_data_vector<int64_t, 2> {
  int4 data;
};
template <>
struct typed_data_vector<float, 2> {
  float2 data;
};
template <>
struct typed_data_vector<float, 4> {
  float4 data;
};
template <>
struct typed_data_vector<int, 2> {
  int2 data;
};
template <>
struct typed_data_vector<int, 4> {
  int4 data;
};//WARNING: A BUG MAYBE
template <>
struct typed_data_vector<int16_t, 2> {
  int data;
};
template <>
struct typed_data_vector<int16_t, 4> {
  int2 data;
};
template <>
struct typed_data_vector<int16_t, 8> {
  int4 data;
};
template <>
struct typed_data_vector<int8_t, 2> {
  int16_t data;
};
template <>
struct typed_data_vector<int8_t, 4> {
  int data;
};
template <>
struct typed_data_vector<int8_t, 8> {
  int2 data;
};
template <>
struct typed_data_vector<int8_t, 16> {
  int4 data;
};
template <typename DataTypeT, int DATA_SIZE>
__device__ __forceinline__ DataTypeT& typed_data_vector_at(
  typed_data_vector<DataTypeT, DATA_SIZE>& v, int idx)
{
  return ((DataTypeT*)(&v.data))[idx];
}

template <>
__device__ __forceinline__ void mov_data<1>(void* to, const void* from)
{
  mov_typed_data(static_cast<int8_t*>(to), static_cast<const int8_t*>(from));
}
template <>
__device__ __forceinline__ void mov_data<2>(void* to, const void* from)
{
  mov_typed_data(static_cast<int16_t*>(to), static_cast<const int16_t*>(from));
}
template <>
__device__ __forceinline__ void mov_data<4>(void* to, const void* from)
{
  mov_typed_data(static_cast<int32_t*>(to), static_cast<const int32_t*>(from));
}
template <>
__device__ __forceinline__ void mov_data<8>(void* to, const void* from)
{
  mov_typed_data(static_cast<int64_t*>(to), static_cast<const int64_t*>(from));
}
template <>
__device__ __forceinline__ void mov_data<16>(void* to, const void* from)
{
  mov_typed_data(static_cast<int4*>(to), static_cast<const int4*>(from));
}

template <typename DataTypeT>
class type_caster {
 public:
  using LoadTypeT  = DataTypeT;
  using StoreTypeT = DataTypeT;
  static __device__ __forceinline__ LoadTypeT convert_load_data(DataTypeT data)
  {
    return static_cast<LoadTypeT>(data);
  }
  static __device__ __forceinline__ DataTypeT convert_store_data(StoreTypeT data)
  {
    return static_cast<DataTypeT>(data);
  }
};
template <>
class type_caster<__half> {
 public:
  using LoadTypeT  = float;
  using StoreTypeT = float;
  static __device__ __forceinline__ LoadTypeT convert_load_data(__half data)
  {
    return static_cast<LoadTypeT>(data);
  }
  static __device__ __forceinline__ __half convert_store_data(StoreTypeT data)
  {
    return static_cast<__half>(data);
  }
};


template <typename FromT, typename ToT>
__device__ __forceinline__ ToT convert_type(FromT from)
{
  return type_caster<ToT>::convert_store_data(type_caster<FromT>::convert_load_data(from));
}

/**
 * Determine alignment of a WholeMemory matrix, in element count, maximum 16 / element_size.
 * @param embedding_desc : wholememory_matrix_description_t matrix description.
 * @return : Alignment that can be used, in element count.
 */
inline int determine_wholememory_alignment_elt_count(desc embedding_desc)
{
  int elt_size = static_cast<int>(sizeof(EmbeddingT));
  assert(elt_size != -1);
  int alignment = 16 / elt_size;
  for (; alignment > 1; alignment /= 2) {
    if (embedding_desc.start_off % alignment == 0 &&
        embedding_desc.dim % alignment == 0 && embedding_desc.stride % alignment == 0)
      break;
  }
  return alignment;
}

/**
 * Determine alignment of normal memory, in element count, maximum 16 / element_size.
 * @param ptr : pointer to the memory.
 * @param memory_desc : wholememory_matrix_description_t matrix description.
 * @return : Alignment that can be used, in element count.
 */
inline int determine_memory_alignment_elt_count(const void* ptr, desc memory_desc)
{
  int elt_size = static_cast<int>(sizeof(InputT));
  assert(elt_size != -1);
  int alignment   = 16 / elt_size;
  int64_t int_ptr = reinterpret_cast<int64_t>(ptr);
  assert(int_ptr % elt_size == 0);
  int_ptr /= elt_size;
  int_ptr += memory_desc.start_off;
  for (; alignment > 1; alignment /= 2) {
    if (int_ptr % alignment == 0 && memory_desc.dim % alignment == 0 &&
        memory_desc.stride % alignment == 0)
      break;
  }
  return alignment;
}

template <int ALIGNMENT = 3>
__global__ void scatter_func_kernel(const InputT* input,
                                    desc input_desc,
                                    const IndexT* indices,
                                    int indice_count,
                                    EmbeddingT* embedding,
                                    desc embedding_desc)
{
  int64_t input_idx          = static_cast<int64_t>(blockIdx.x) * blockDim.y + threadIdx.y;
  int thread_idx             = threadIdx.x;
  IndexT embedding_table_idx = indices[input_idx];
  if (embedding_table_idx < 0) return;
  //wholememory::device_reference<EmbeddingT> embedding_dev_ref(embedding_gref);
  int embedding_size       = embedding_desc.dim;
  int64_t embedding_stride = embedding_desc.stride;
  int64_t input_stride     = input_desc.stride;
  typed_data_vector<EmbeddingT, ALIGNMENT> embeddings;
  typed_data_vector<InputT, ALIGNMENT> inputs;
  const InputT* input_ptr  = input + input_desc.start_off + input_stride * input_idx;
  int64_t embedding_offset = embedding_desc.start_off + embedding_table_idx * embedding_stride;
  for (; input_idx < indice_count; input_idx += static_cast<int64_t>(gridDim.x) * blockDim.y) {
    for (int emb_idx = thread_idx * ALIGNMENT; emb_idx < embedding_size; emb_idx += ALIGNMENT * blockDim.x) {
      mov_data<sizeof(InputT) * ALIGNMENT>(&inputs, input_ptr + emb_idx);
#pragma unroll
      for (int sub_idx = 0; sub_idx < ALIGNMENT; sub_idx++) {
        typed_data_vector_at(embeddings, sub_idx) =
          convert_type<InputT, EmbeddingT>(typed_data_vector_at(inputs, sub_idx));
      }
      mov_data<sizeof(EmbeddingT) * ALIGNMENT>(embedding + embedding_offset + emb_idx,
                                               &embeddings);
    }
  }
}

#define shm_size (16384/sizeof(EmbeddingT))//TODO this may be important
//#define shm_size (4096/sizeof(EmbeddingT))//TODO this may be important
template <int ALIGNMENT = 3>
__global__ void scatter_kernel_TMA(const InputT* input,
                                    desc input_desc,
                                    const IndexT* indices,
                                    int indice_count,
                                    EmbeddingT* embedding,
                                    desc embedding_desc)
{
  auto grid = cooperative_groups::this_grid();
  auto block = cooperative_groups::this_thread_block();
  extern __shared__ EmbeddingT shared[];
  int embedding_size       = embedding_desc.dim;
  int64_t embedding_stride = embedding_desc.stride;
  int block_idx = block.group_index().x;
  int64_t input_stride     = input_desc.stride;
  int batch_size = shm_size/input_stride;//indices batch size in lines
  typed_data_vector<EmbeddingT, ALIGNMENT> embeddings;
  typed_data_vector<InputT, ALIGNMENT> inputs;
  for (int64_t input_idx = block_idx*batch_size; input_idx < indice_count; input_idx += grid.num_blocks()*batch_size) {
	  int cur_idx_lines = (indice_count - input_idx) > batch_size ? batch_size : indice_count - input_idx;
	  const InputT* input_ptr = input + input_desc.start_off + input_stride * input_idx;
	  cooperative_groups::memcpy_async(block, shared, (EmbeddingT*)(input_ptr), sizeof(EmbeddingT)*cur_idx_lines*input_stride);
	  cooperative_groups::wait(block);
	  for (int e = 0; e < cur_idx_lines; e ++) {
		  int64_t embedding_table_idx = indices[input_idx + e];
	  	EmbeddingT *emb_ptr = embedding + embedding_desc.start_off + embedding_table_idx*embedding_stride;
      
      for (int emb_idx = block.thread_rank() * ALIGNMENT; emb_idx < embedding_size; emb_idx += ALIGNMENT * block.size()) {
        mov_data<sizeof(InputT) * ALIGNMENT>(&inputs, shared +e*input_stride + emb_idx);
#pragma unroll
        for (int sub_idx = 0; sub_idx < ALIGNMENT; sub_idx++) {
          typed_data_vector_at(embeddings, sub_idx) =
            convert_type<InputT, EmbeddingT>(typed_data_vector_at(inputs, sub_idx));
        }
        mov_data<sizeof(EmbeddingT) * ALIGNMENT>(emb_ptr + emb_idx, &embeddings);
      }
	  }
    block.sync();
  }
  return ;
}
template<int ALIGNMENT = 3,uint8_t stage_count = 2>//TODO set stage count to 2
__global__ void scatter_kernel_TMA_pipeline(const InputT* input,
                                    desc input_desc,
                                    const IndexT* indices,
                                    int indice_count,
                                    EmbeddingT* embedding,
                                    desc emb_desc)
{
  //auto grid = cooperative_groups::this_grid();
  auto block = cooperative_groups::this_thread_block();

  extern __shared__ EmbeddingT shared[]; // stages_count * block.size() * sizeof(int) bytes
  
  __shared__ cuda::pipeline_shared_state<cuda::thread_scope::thread_scope_block,stage_count> shared_state;
  auto pipeline = cuda::make_pipeline(block, &shared_state);

  int input_stride = input_desc.stride;
  int batch_size = shm_size/stage_count/input_stride;
  int block_idx = block.group_index().x;
  int my_batch_num = ((indice_count+batch_size-1)/batch_size) % gridDim.x > block_idx ? 1 : 0;
  my_batch_num += indice_count/(batch_size*gridDim.x);
  
  typed_data_vector<EmbeddingT, ALIGNMENT> embeddings;
  typed_data_vector<InputT, ALIGNMENT> inputs;

  for (int get_batch = 0, put_batch = 0; put_batch < my_batch_num; put_batch ++) {
    for (; get_batch < my_batch_num && get_batch < (put_batch + stage_count); get_batch ++) {
      int64_t input_idx = (get_batch * gridDim.x + block_idx) * batch_size;
      int idx_line_num = (indice_count - input_idx) > batch_size ? batch_size : (indice_count - input_idx);
      int shared_off = (get_batch % stage_count)*batch_size*input_stride;
      EmbeddingT *in_addr = (EmbeddingT*)(input + input_desc.start_off + input_stride * input_idx);
      pipeline.producer_acquire();
      cuda::memcpy_async(block, shared+shared_off, in_addr, sizeof(EmbeddingT)*idx_line_num*input_stride, pipeline);
      pipeline.producer_commit();
    }
    int64_t input_idx = (put_batch * gridDim.x + block_idx) * batch_size;
    int idx_line_num = (indice_count - input_idx) > batch_size ? batch_size : (indice_count - input_idx);
    pipeline.consumer_wait();
    for (int e = 0; e < idx_line_num; e ++) {
      int64_t emb_idx = indices[input_idx+e];
      EmbeddingT* out_addr = embedding + emb_desc.start_off + emb_desc.stride*emb_idx;
      int shared_off = (put_batch % stage_count)* batch_size*input_stride;
      /*for (int emb_idx = block.thread_rank(); emb_idx < emb_desc.dim; emb_idx += block.size()) {
        out_addr[emb_idx] = shared[shared_off+e*input_stride+emb_idx];
      }*/
      for (int emb_idx = block.thread_rank() * ALIGNMENT; emb_idx < emb_desc.dim; emb_idx += ALIGNMENT * block.size()) {
        mov_data<sizeof(InputT) * ALIGNMENT>(&inputs, shared + shared_off + e*input_stride + emb_idx);
#pragma unroll
        for (int sub_idx = 0; sub_idx < ALIGNMENT; sub_idx++) {
          typed_data_vector_at(embeddings, sub_idx) =
            convert_type<InputT, EmbeddingT>(typed_data_vector_at(inputs, sub_idx));
        }
        mov_data<sizeof(EmbeddingT) * ALIGNMENT>(out_addr + emb_idx, &embeddings);
      }
    }
    //block.sync();
    pipeline.consumer_release();
    block.sync();
  }
  return ;
}

void scatter_temp_func(InputT* input,
                       struct desc input_desc,
                       IndexT* indices,
                       int indice_count,
                       EmbeddingT* embedding,
                       struct desc embedding_desc)
{
  if (indice_count == 0 || embedding_desc.dim == 0) return;
  int wm_alignment   = determine_wholememory_alignment_elt_count(embedding_desc);
  int mm_alignment   = determine_memory_alignment_elt_count(input, input_desc);
  int alignment      = std::min<int>(wm_alignment, mm_alignment);
  int embedding_size = embedding_desc.dim;
#ifdef NAIVE_COPY
  int thread_x       = (embedding_size + alignment-1)/alignment;
  thread_x           = std::min(thread_x, 256);
  int thread_y       = 1;
  if (thread_x < 64) {
    int power2_thread_x = 1;
    for (; power2_thread_x < thread_x; power2_thread_x *= 2)
      ;
    thread_x = power2_thread_x;
    thread_y = 64 / thread_x;
  }
  int64_t block_count_64 = (indice_count + thread_y - 1) / thread_y;
  int block_count = block_count_64 >= INT_MAX ? INT_MAX / 4 : static_cast<int>(block_count_64);
  dim3 block_dim(thread_x, thread_y, 1);
  void (*kernel_fn)(const InputT*,
                    desc,
                    const IndexT*,
                    int,
                    EmbeddingT*,
                    desc) = nullptr;
  //printf("key parameters: %d %d %d %d\n",thread_x, thread_y, block_count, alignment);
  switch (alignment) {
    case 16: {kernel_fn = scatter_func_kernel<16>;break;}
    case 8: {kernel_fn = scatter_func_kernel<8>;break;}
    case 4: {kernel_fn = scatter_func_kernel<4>;break;}
    case 2: {kernel_fn = scatter_func_kernel<2>;break;}
    case 1: {kernel_fn = scatter_func_kernel<1>;break;}
    default: {
      printf("scatter func alignment=%d.\n", alignment);
      return;
    }
  }
#endif
  hipEvent_t start, stop;
	float esp_time_gpu;
	CUDA_TRY(hipEventCreate(&start));
	CUDA_TRY(hipEventCreate(&stop));
  CUDA_TRY(hipEventRecord(start, 0));
#ifdef NAIVE_COPY
  kernel_fn<<<block_count, block_dim>>>(input,
                                        input_desc,
                                        indices,
                                        indice_count,
                                        embedding,
                                        embedding_desc);
#endif
#ifdef TMA_COPY
  void (*kernel_fn)(const InputT*,
                    desc,
                    const IndexT*,
                    int,
                    EmbeddingT*,
                    desc) = nullptr;
  //printf("key parameters: %d %d %d %d\n",thread_x, thread_y, block_count, alignment);
  switch (alignment) {
    case 16: {  kernel_fn = scatter_kernel_TMA<16>; break;}
    case 8: {  kernel_fn = scatter_kernel_TMA<8>; break;}
    case 4: {  kernel_fn = scatter_kernel_TMA<4>; break;}
    case 2: {  kernel_fn = scatter_kernel_TMA<2>; break;}
    case 1: {  kernel_fn = scatter_kernel_TMA<1>; break;}
    default: {
      printf("scatter func alignment=%d.\n", alignment); return;
    }
  }
  int block_size = (embedding_desc.dim + alignment-1)/alignment;
  block_size = block_size > 512 ? 512 : block_size;
  int block_count = indice_count > 1024 ? 1024 : indice_count;
  kernel_fn<<<block_count, block_size, shm_size*sizeof(EmbeddingT)>>>(input,
                                                                          input_desc,
                                                                          indices,
                                                                          indice_count,
                                                                          embedding,
                                                                          embedding_desc);
#endif
#ifdef TMA_PIPELINE_COPY
   void (*kernel_fn)(const InputT*,
                    desc,
                    const IndexT*,
                    int,
                    EmbeddingT*,
                    desc) = nullptr;
  //printf("key parameters: %d %d %d %d\n",thread_x, thread_y, block_count, alignment);
  switch (alignment) {
    case 16: {  kernel_fn = scatter_kernel_TMA_pipeline<16,2>; break;}
    case 8: {  kernel_fn = scatter_kernel_TMA_pipeline<8,2>; break;}
    case 4: {  kernel_fn = scatter_kernel_TMA_pipeline<4,2>; break;}
    case 2: {  kernel_fn = scatter_kernel_TMA_pipeline<2,2>; break;}
    case 1: {  kernel_fn = scatter_kernel_TMA_pipeline<1,2>; break;}
    default: {
      printf("scatter func alignment=%d.\n", alignment); return;
    }
  }
  int block_size = (embedding_desc.dim + alignment-1)/alignment;
  block_size = block_size > 256 ? 256 : block_size;
  int block_count = indice_count > 4096 ? 4096 : indice_count;
  kernel_fn<<<block_count, block_size, shm_size*sizeof(EmbeddingT)>>>(input,
                                                                   input_desc,
                                                                   indices,
                                                                   indice_count,
                                                                   embedding,
                                                                   embedding_desc);

#endif
  CUDA_TRY(hipDeviceSynchronize());
  CUDA_TRY(hipEventRecord(stop, 0));
	CUDA_TRY(hipEventSynchronize(stop));
  CUDA_TRY(hipEventElapsedTime(&esp_time_gpu, start, stop));
	printf("    Time for the kernel is: %f ms, where alignment is %d\n", esp_time_gpu, alignment);
  return ;
}

int main (int argc, char**argv) {
  //key parameters
  int embedding_dim = 128;
  if (argc > 1) embedding_dim = atoi(argv[1]);
  int emb_start_off = 0;//the offset is also in element
  if (argc > 2) emb_start_off = atoi(argv[2]);
  int input_start_off = 0;//emb_start_off;
  if (argc > 3) input_start_off = atoi(argv[3]);
  int input_dim = embedding_dim;
  uint64_t embedding_size = 10 * 1024UL * 1024UL;
  uint64_t input_size = 5 * 1024UL * 1024UL;

  uint64_t total_size_gb = (embedding_size + input_size)*embedding_dim*sizeof(EmbeddingT)/1024/1024/1024;
  printf("the total size is %d GB\n", total_size_gb);
  //construct input
  InputT *input;
  int in_aligned_size = 16/sizeof(InputT);
  int in_stride = input_dim % in_aligned_size == 0 ? 
                     input_dim : (input_dim/in_aligned_size+1)*in_aligned_size;
  int64_t in_malloc_size = (int64_t)in_stride * input_size + input_start_off;
  CUDA_TRY(hipMalloc((void **)&input, sizeof(InputT)*in_malloc_size));
  printf("the input stride is %d, the input_malloc_size is %ld\n", in_stride, in_malloc_size);

  thrust::sequence(thrust::device, input+input_start_off, input+in_malloc_size, 0);//NOTE: more initialization methods needed
  thrust::reverse(thrust::device, input+input_start_off, input+in_malloc_size);
  struct desc input_desc = desc(input_size, input_dim, in_stride, input_start_off);
  printf("construct input tensor done, the in_stride is %d\n", in_stride);

  //construct embedding
  EmbeddingT * embedding;
  int emb_aligned_size = 16/sizeof(EmbeddingT);
  int emb_stride = embedding_dim % emb_aligned_size == 0 ? 
                     embedding_dim : (embedding_dim/emb_aligned_size+1)*emb_aligned_size;
  int64_t emb_malloc_size = (int64_t)emb_stride * embedding_size + emb_start_off;
  printf("the emb stride is %d, the emb_malloc_size is %ld\n", emb_stride, emb_malloc_size);
  CUDA_TRY(hipMalloc((void **)&embedding, sizeof(EmbeddingT)*emb_malloc_size));
  thrust::sequence(thrust::device, embedding+emb_start_off, embedding+emb_malloc_size, 0);
  struct desc emb_desc = desc(embedding_size, embedding_dim, emb_stride, emb_start_off);
  printf("construct the target embedding done, the emb_stride is %d\n", emb_stride);

  //construct indices
  IndexT *indices;
  CUDA_TRY(hipMalloc((void **)&indices, sizeof(IndexT)*input_size));
  IndexT *h_indices = (IndexT*)malloc(sizeof(IndexT)*input_size);
#ifdef QUICK_VALIDATION
  EmbeddingT* h_embedding = (EmbeddingT*)malloc(sizeof(EmbeddingT)*emb_malloc_size);
  InputT* h_input = (InputT *)malloc(sizeof(InputT)*in_malloc_size);
#endif
  uint8_t* used = (uint8_t*)malloc(sizeof(uint8_t)*embedding_size); 
  for (int iter = 0; iter < 2; iter ++) {
    printf("    start generating the indices for %d th iteration...\n", iter);
    //different iterations have different indices
    int min = 0, max = embedding_size-1;
    random_device seed;
	  ranlux48 engine(seed());
    uniform_int_distribution<> distrib(min, max);
    memset(used, 0, sizeof(uint8_t)*embedding_size);
    for (int i = 0; i < input_size; i ++) {
      int random = distrib(engine);//随机数
      while(used[random]) {
        random = distrib(engine);
      }
      used[random] = 1;
      //NOTE: currently only int/half_int is supported
      h_indices[i] = (IndexT)random;
    }
    thrust::sort(thrust::host, h_indices, h_indices + input_size);
    CUDA_TRY(hipMemcpy(indices, h_indices, sizeof(IndexT)*input_size, hipMemcpyHostToDevice));
    CUDA_TRY(hipDeviceSynchronize());
    printf("    indices prepared, start the scatter function now...\n");
    scatter_temp_func(input,
                      input_desc,
                      indices,
                      input_size,
                      embedding,
                      emb_desc);
#ifdef QUICK_VALIDATION//NOTE the check here is designed for int type
    CUDA_TRY(hipMemcpy(h_embedding, embedding, sizeof(EmbeddingT)*emb_malloc_size, hipMemcpyDeviceToHost));
    CUDA_TRY(hipMemcpy(h_indices, indices, sizeof(IndexT)*input_size, hipMemcpyDeviceToHost));
    CUDA_TRY(hipMemcpy(h_input, input, sizeof(InputT)*in_malloc_size, hipMemcpyDeviceToHost));
    CUDA_TRY(hipDeviceSynchronize());

    /*printf("the embedding table:\n");
    for (uint64_t i = 0; i < embedding_size; i ++) {
      for (int j = 0; j < embedding_dim; j ++)
        printf("%d ", h_embedding[i*emb_stride+j+emb_start_off]);
      printf("\n");
    }
    printf("the input table:\n");
    for (uint64_t i = 0; i < input_size; i ++) {
      for (int j = 0; j < input_dim; j ++)
        printf("%d ", h_input[i*in_stride+j+input_start_off]);
      printf("\n");
    }
    printf("the indices:\n");
    for (int i = 0; i < input_size; i ++)
      printf("%d ", h_indices[i]);
    printf("\n");*/

    int index_pos = 0;
    //int index_print_step = (input_size/100 > 1) ? input_size/100 : 1;
    bool valid = true;
    for (uint64_t i = 0; i < embedding_size; i ++) {
      if (i == h_indices[index_pos]) {
        //if (index_pos % index_print_step == 0)
          //printf("the %d th index for %d th iteration is %ld\n", index_pos, iter, i);
        if (h_embedding[i*emb_stride + emb_start_off] != in_malloc_size-input_start_off-1-index_pos*in_stride) {
          valid = false;
          printf("scattered, i = %lu, index_pos = %d, embedding ele is %d, should be %d\n",
                                i, index_pos, h_embedding[i*emb_stride + emb_start_off], 
                                in_malloc_size-input_start_off-1-index_pos*in_stride);
          break;
        }
        index_pos ++;
      } else {
        if (h_embedding[i*emb_stride + emb_start_off] != i*emb_stride) {
          valid = false;
          printf("original, i = %lu, embedding ele is %d, should be %lu\n",
                            i, h_embedding[i*emb_stride + emb_start_off], i*emb_stride);
          break;
        }
      }
    }
    if (valid)
      printf("    the %d th iteration passed the quick validation!\n", iter);
    else 
      printf("    the %d th iteration didn't pass!\n", iter);
#endif
    thrust::sequence(thrust::device, embedding+emb_start_off, embedding+emb_malloc_size, 0);
    printf("\n");
  }
#ifdef QUICK_VALIDATION
  free(h_embedding);
  free(h_input);
#endif
  free(used);
  free(h_indices);
  CUDA_TRY(hipFree(input));
  CUDA_TRY(hipFree(embedding));
  CUDA_TRY(hipFree(indices));
  printf("exit now\n");
  return 0;
}
